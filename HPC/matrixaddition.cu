#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void addition(float *A, float *B, float *C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    int N = 4; // Corrected from 2 to 4
    size_t size = N * sizeof(float);
    float A[] = {1, 2, 3, 4};
    float B[] = {5, 6, 7, 8};
    float C[4];

    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 threads(256);
    dim3 blocks((N + threads.x - 1) / threads.x);

    addition<<<blocks, threads>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Print Result
    printf("Resultant Vector C:\n");
    for (int i = 0; i < N; i++)
    {
        printf("%f ", C[i]);
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
