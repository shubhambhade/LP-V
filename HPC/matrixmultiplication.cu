#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matmul(float *A, float *B, float *C, int N)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N)
    {
        float sum = 0;
        for (int k = 0; k < N; k++)
        {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main()
{
    int N = 2;
    size_t size = N * N * sizeof(float);
    float A[] = {1, 2, 3, 4};
    float B[] = {5, 6, 7, 8};
    float C[4];

    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 blocks(N, N);
    dim3 threads(1, 1);

    matmul<<<blocks, threads>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Print Result
    printf("Resultant Matrix C:\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%f ", C[i * N + j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
